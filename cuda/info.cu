#include <cstdio>
#include<hip/hip_runtime.h>

extern "C" 
int getDeviceName_main(char* name, int device)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    sprintf(name, "%s", prop.name);
    return 0;
}

extern "C"
int getTotalMem_main(size_t* mem, int device)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    *mem = prop.totalGlobalMem;
    return 0;
}
