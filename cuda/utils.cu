#include <cstdio>
#include<hip/hip_runtime.h>

extern "C" 
int getDeviceName_main(char* name, int device)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    sprintf(name, "%s", prop.name);
    return 0;
}
