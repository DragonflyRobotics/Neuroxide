#include <hip/hip_runtime.h>
#include <math.h>

extern "C" __global__ void math_operations(float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        output[idx] = sinf(input[idx]) + cosf(input[idx]) + sqrtf(input[idx]);
    }
}

extern "C" void launch_math_operations(float* input, float* output, int N) {
    float* d_input;
    float* d_output;

    hipMalloc((void**)&d_input, N * sizeof(float));
    hipMalloc((void**)&d_output, N * sizeof(float));

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;

    math_operations<<<blocks_per_grid, threads_per_block>>>(d_input, d_output, N);

    hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

